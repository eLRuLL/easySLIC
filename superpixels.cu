#include "hip/hip_runtime.h"
#define GL_GLEXT_PROTOTYPES
#include <GL/glut.h>
#define GLM_FORCE_RADIANS
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtx/norm.hpp>
#include <glm/gtx/vector_angle.hpp>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <iostream>
#include <vector>

#include <stdio.h>
#include <float.h>

#include "superpixels.h"

hipGraphicsResource *in_resource;
hipGraphicsResource *out_resource;
uchar3* in_image;
uchar3* out_image;

__constant__ int img_width;
__constant__ int img_height;

__device__ int2 find_local_minimum( uchar3 *image, int2 center, int index ){
    int i,j;
    float min_grad = FLT_MAX;
    int2 loc_min = center;

    for (i = center.x-1; i < center.x+2; ++i) {
        for (j = center.y-1; j < center.y+2; ++j) {
          /* get L values. */
          unsigned char i1 = image[index+1].x;
          unsigned char i2 = image[index+img_width].x;
          unsigned char i3 = image[index].x;

          /* Compute horizontal and vertical gradients and keep track of the
          minimum. */
          if (sqrtf(powf(i1 - i3, 2)) + sqrtf(powf(i2 - i3,2)) < min_grad) {
            min_grad = fabsf(i1 - i3) + fabsf(i2 - i3);
            loc_min.x = i;
            loc_min.y = j;
          }
        }
      }

      return loc_min;
}

// __global__ void init_data( uchar3* image ){
//     int i,j;
// }

// __global__ void generate_superpixels(){

// }

__global__ void redkernel( uchar3 *in_image, uchar3 *out_image ) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    out_image[offset].x = 255;
    out_image[offset].y = 0;
    out_image[offset].z = 0;

}

__global__ void GPU_invert( uchar3 *in_image, uchar3 *out_image ) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	out_image[offset].x = 255 - in_image[offset].x;
	out_image[offset].y = 255 - in_image[offset].y;
	out_image[offset].z = 255 - in_image[offset].z;

}

void interop_setup(int M, int N) {
	hipDeviceProp_t prop;
	int dev;
	memset( &prop, 0, sizeof( hipDeviceProp_t ) );
	prop.major = 1;
	prop.minor = 0;
	hipChooseDevice( &dev, &prop );
	cudaGLSetGLDevice( dev );	// dev = 0
	hipMemcpyToSymbol(HIP_SYMBOL(img_width),&M,1*sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(img_height),&N,1*sizeof(int),0,hipMemcpyHostToDevice);
}

void interop_register_buffer(GLuint& in_buffer, GLuint& out_buffer){
	hipGraphicsGLRegisterBuffer( &in_resource,in_buffer,cudaGraphicsMapFlagsNone);
	hipGraphicsGLRegisterBuffer( &out_resource,out_buffer,cudaGraphicsMapFlagsNone);
}

void interop_map() {
	size_t in_size, out_size;

	hipGraphicsMapResources( 1, &in_resource, NULL );
	hipGraphicsResourceGetMappedPointer( (void**)&in_image,&in_size,in_resource ) ;

	hipGraphicsMapResources( 1, &out_resource, NULL );
	hipGraphicsResourceGetMappedPointer( (void**)&out_image,&out_size,out_resource ) ;
}

void interop_run(int M, int N) {

	dim3 grids(N,M);
	dim3 threads(1,1);

	redkernel<<<grids,threads>>>( in_image, out_image);
	hipGraphicsUnmapResources( 1, &in_resource, NULL );
	hipGraphicsUnmapResources( 1, &out_resource, NULL );

}

void interop_cleanup(){
	hipGraphicsUnregisterResource( in_resource );
	hipGraphicsUnregisterResource( out_resource );
}